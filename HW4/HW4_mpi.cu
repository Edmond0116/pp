#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <mpi.h>
#include <hip/hip_runtime.h>
#define INF 1000000000
#define V 20001
#define ceil(a, b) (1 + (((a) - 1) / (b)))
#define min(a, b) (((a) < (b)) ? (a) : (b))

// <<<1, dim3(B, B)>>>
template<int B>
__global__ void phase1(int n, int r, int *deviceDist, size_t pitch) {
    const int kmin = r * B,
        kmax = min(B, n - kmin),
        i = threadIdx.y,
        j = threadIdx.x,
        ni = kmin + i,
        nj = kmin + j;
    int t, v;
    if (ni >= n || nj >= n) return;
    __shared__ int sharedDist[B][B];
    v = sharedDist[i][j] = *((int *)((char *) deviceDist + ni * pitch) + nj);
    #pragma unroll
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        if ((t = sharedDist[i][k] + sharedDist[k][j]) < v)
            v = sharedDist[i][j] = t;
    }
    *((int *)((char *) deviceDist + ni * pitch) + nj) = v;
}
// <<<dim3(2, round), dim3(B, B)>>>
template<int B>
__global__ void phase2(int n, int r, int *deviceDist, size_t pitch) {
    if (blockIdx.y == r) return;
    const int kmin = r * B,
        kmax = min(B, n - kmin),
        i = threadIdx.y,
        j = threadIdx.x,
        ni = (blockIdx.x == 0 ? blockIdx.y : r) * B + i,
        nj = (blockIdx.x == 0 ? r : blockIdx.y) * B + j,
        ki = kmin + i,
        kj = kmin + j;
    __shared__ int sharedDist[B][B], pivotDist[B][B];
    int t, v,
        (*fromDist)[B] = blockIdx.x == 0 ? sharedDist : pivotDist,
        (*toDist)[B] = blockIdx.x == 0 ? pivotDist : sharedDist;
    v = sharedDist[i][j] = ni >= n || nj >= n ? INF :
        *((int *)((char *) deviceDist + ni * pitch) + nj);
    pivotDist[i][j] = ki >= n || kj >= n ? INF :
        *((int *)((char *) deviceDist + ki * pitch) + kj);
    if (ni >= n || nj >= n) return;
    #pragma unroll
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        if ((t = fromDist[i][k] + toDist[k][j]) < v)
            v = sharedDist[i][j] = t;
    }
    *((int *)((char *) deviceDist + ni * pitch) + nj) = v;
}
// <<<dim3(round, round), dim3(B, B)>>>
template<int B>
__global__ void phase3(int n, int r, int *deviceDist, size_t pitch, int ompIdx) {
    if (blockIdx.x == r || blockIdx.y + ompIdx == r) return;
    const int kmin = r * B,
        kmax = min(B, n - kmin),
        i = threadIdx.y,
        j = threadIdx.x,
        ni = (blockIdx.y + ompIdx) * B + i,
        nj = blockIdx.x * B + j,
        ki = kmin + i,
        kj = kmin + j;
    __shared__ int vertDist[B][B], horzDist[B][B];
    int t, v;
    vertDist[i][j] = ni >= n || kj >= n ? INF :
        *((int *)((char *) deviceDist + ni * pitch) + kj);
    horzDist[i][j] = ki >= n || nj >= n ? INF :
        *((int *)((char *) deviceDist + ki * pitch) + nj);
    if (ni >= n || nj >= n) return;
    v = *((int *)((char *) deviceDist + ni * pitch) + nj);
    __syncthreads();
    #pragma unroll
    for (int k = 0; k < kmax; ++k) {
        if ((t = vertDist[i][k] + horzDist[k][j]) < v)
            v = t;
    }
    *((int *)((char *) deviceDist + ni * pitch) + nj) = v;
}

int main(int argc, char* argv[]) {
    int rank;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    // input
    int n, m, a, b, v, *Dist, *deviceDist[2];
    std::ifstream infile(argv[1]);
    infile >> n >> m;
    hipHostMalloc((void **) &Dist, n * n * sizeof(int));
    for (size_t i = 0; i < n; ++i) {
        for (size_t j = 0; j < n; ++j) {
            Dist[i * n + j] = (i == j ? 0 : INF);
        }
    }
    while (m--) {
        infile >> a >> b >> v;
        Dist[a * n + b] = v;
    }
    infile.close();
    // blocked FW
    size_t pitch[2];
    const int B = atoi(argv[3]), round = ceil(n, B),
            rhalf = round / 2;
    dim3 thread(B, B);
    hipSetDevice(rank);
    hipMallocPitch(&deviceDist[rank], &pitch[rank], n * sizeof(int), n);
    hipMemcpy2D(deviceDist[rank], pitch[rank], Dist, n * sizeof(int), n * sizeof(int), n, hipMemcpyHostToDevice);
    MPI_Request req;
    const int rblock = rank == 0 ? (round + 1) / 2 : rhalf;
    int kmin = 0, kmax;
    phase1<32><<<1, thread>>>(n, 0, deviceDist[rank], pitch[rank]);
    phase2<32><<<dim3(2, round), thread>>>(n, 0, deviceDist[rank], pitch[rank]);
    phase3<32><<<dim3(round, rblock), thread>>>(n, 0, deviceDist[rank], pitch[rank], rank == 0 ? rhalf : 0);
    for (int r = 1; r < round; ++r) {
        kmin += B, kmax = min(B, n - kmin);
        if (rank == 0) {
            MPI_Irecv(Dist + kmin, rhalf * B * kmax, MPI_INT, 1, 0, MPI_COMM_WORLD, &req);
            hipMemcpy2D(Dist + rhalf * B * n + kmin, n * sizeof(int),
                (char *) deviceDist[0] + rhalf * B * pitch[0] + kmin * sizeof(int), pitch[0],
                kmax, n - rhalf * B, hipMemcpyDeviceToHost);
            MPI_Send(Dist + rhalf * B * n + kmin, (n - rhalf * B) * kmax, MPI_INT, 1, 0, MPI_COMM_WORLD);
            if (r < rhalf) {
                MPI_Recv(Dist + kmin * n, kmax * n, MPI_INT, 1, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                hipMemcpy2D((char *) deviceDist[0] + kmin * pitch[0], pitch[0],
                    Dist + kmin * n, n * sizeof(int),
                    n * sizeof(int), kmax, hipMemcpyHostToDevice);
            } else {
                hipMemcpy2D(Dist + kmin * n, n * sizeof(int),
                    (char *) deviceDist[0] + kmin * pitch[0], pitch[0],
                    n * sizeof(int), kmax, hipMemcpyDeviceToHost);
                MPI_Send(Dist + kmin * n, kmax * n, MPI_INT, 1, 1, MPI_COMM_WORLD);
            }
            MPI_Wait(&req, MPI_STATUS_IGNORE);
            hipMemcpy2D((char *) deviceDist[0] + kmin * sizeof(int), pitch[0], Dist + kmin, n * sizeof(int),
                kmax, rhalf * B, hipMemcpyHostToDevice);
        } else if (rhalf) {
            MPI_Irecv(Dist + rhalf * B * n + kmin, (n - rhalf * B) * kmax, MPI_INT, 0, 0, MPI_COMM_WORLD, &req);
            hipMemcpy2D(Dist + kmin, n * sizeof(int), (char *) deviceDist[1] + kmin * sizeof(int), pitch[1],
                kmax, rhalf * B, hipMemcpyDeviceToHost);
            MPI_Send(Dist + kmin, rhalf * B * kmax, MPI_INT, 0, 0, MPI_COMM_WORLD);
            if (r >= rhalf) {
                MPI_Recv(Dist + kmin * n, kmax * n, MPI_INT, 0, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                hipMemcpy2D((char *) deviceDist[1] + kmin * pitch[1], pitch[1],
                    Dist + kmin * n, n* sizeof(int),
                    n * sizeof(int), kmax, hipMemcpyHostToDevice);
            } else {
                hipMemcpy2D(Dist + kmin * n, n * sizeof(int),
                    (char *) deviceDist[1] + kmin * pitch[1], pitch[1],
                    n * sizeof(int), kmax, hipMemcpyDeviceToHost);
                MPI_Send(Dist + kmin * n, kmax * n, MPI_INT, 0, 1, MPI_COMM_WORLD);
            }
            MPI_Wait(&req, MPI_STATUS_IGNORE);
            hipMemcpy2D((char *) deviceDist[1] + rhalf * B * pitch[1] + kmin * sizeof(int), pitch[1],
                Dist + rhalf * B * n + kmin, n * sizeof(int),
                kmax, n - rhalf * B, hipMemcpyHostToDevice);
        }
        phase1<32><<<1, thread>>>(n, r, deviceDist[rank], pitch[rank]);
        phase2<32><<<dim3(2, round), thread>>>(n, r, deviceDist[rank], pitch[rank]);
        phase3<32><<<dim3(round, rblock), thread>>>(n, r, deviceDist[rank], pitch[rank], rank == 0 ? rhalf : 0);
    }
    hipDeviceSynchronize();
    // output
    MPI_File outfile;
    MPI_File_open(MPI_COMM_WORLD, argv[2], MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &outfile);
    if (rank == 0) {
        hipMemcpy2D(
            Dist + rhalf * B * n, n * sizeof(int), (char *) deviceDist[0] + rhalf * B * pitch[0], pitch[0],
            n * sizeof(int), n - rhalf * B, hipMemcpyDeviceToHost);
        MPI_File_write_at_all(outfile,
            rhalf * B * n * sizeof(int),
            Dist + rhalf * B * n,
            (n - rhalf * B) * n,
            MPI_INT, MPI_STATUS_IGNORE);
    } else {
        hipMemcpy2D(Dist, n * sizeof(int), deviceDist[1], pitch[1],
            n * sizeof(int), rhalf * B, hipMemcpyDeviceToHost);
        MPI_File_write_at_all(outfile, 0, Dist, rhalf * B * n, MPI_INT, MPI_STATUS_IGNORE);
    }
    hipFree(deviceDist[rank]);
    MPI_Finalize();
    return 0;
}

