#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <hip/hip_runtime.h>
#define INF 1000000000
#define V 20001
#define ceil(a, b) (1 + (((a) - 1) / (b)))
#define min(a, b) (((a) < (b)) ? (a) : (b))

// <<<1, dim3(B, B)>>>
template<int B>
__global__ void phase1(int n, int r, int *deviceDist, size_t pitch) {
    const int kmin = r * B,
        kmax = min(B, n - kmin),
        i = threadIdx.y,
        j = threadIdx.x,
        ni = kmin + i,
        nj = kmin + j;
    int t, v;
    if (ni >= n || nj >= n) return;
    __shared__ int sharedDist[B][B];
    v = sharedDist[i][j] = *((int *)((char *) deviceDist + ni * pitch) + nj);
    #pragma unroll
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        if ((t = sharedDist[i][k] + sharedDist[k][j]) < v)
            v = sharedDist[i][j] = t;
    }
    *((int *)((char *) deviceDist + ni * pitch) + nj) = v;
}
// <<<dim3(2, round), dim3(B, B)>>>
template<int B>
__global__ void phase2(int n, int r, int *deviceDist, size_t pitch) {
    if (blockIdx.y == r) return;
    const int kmin = r * B,
        kmax = min(B, n - kmin),
        i = threadIdx.y,
        j = threadIdx.x,
        ni = (blockIdx.x == 0 ? blockIdx.y : r) * B + i,
        nj = (blockIdx.x == 0 ? r : blockIdx.y) * B + j,
        ki = kmin + i,
        kj = kmin + j;
    __shared__ int sharedDist[B][B], pivotDist[B][B];
    int t, v,
        (*fromDist)[B] = blockIdx.x == 0 ? sharedDist : pivotDist,
        (*toDist)[B] = blockIdx.x == 0 ? pivotDist : sharedDist;
    v = sharedDist[i][j] = ni >= n || nj >= n ? INF :
        *((int *)((char *) deviceDist + ni * pitch) + nj);
    pivotDist[i][j] = ki >= n || kj >= n ? INF :
        *((int *)((char *) deviceDist + ki * pitch) + kj);
    if (ni >= n || nj >= n) return;
    #pragma unroll
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        if ((t = fromDist[i][k] + toDist[k][j]) < v)
            v = sharedDist[i][j] = t;
    }
    *((int *)((char *) deviceDist + ni * pitch) + nj) = v;
}
// <<<dim3(round, round), dim3(B, B)>>>
template<int B>
__global__ void phase3(int n, int r, int *deviceDist, size_t pitch) {
    if (blockIdx.x == r || blockIdx.y == r) return;
    const int kmin = r * B,
        kmax = min(B, n - kmin),
        i = threadIdx.y,
        j = threadIdx.x,
        ni = blockIdx.y * B + i,
        nj = blockIdx.x * B + j,
        ki = kmin + i,
        kj = kmin + j;
    __shared__ int vertDist[B][B], horzDist[B][B];
    int t, v;
    vertDist[i][j] = ni >= n || kj >= n ? INF :
        *((int *)((char *) deviceDist + ni * pitch) + kj);
    horzDist[i][j] = ki >= n || nj >= n ? INF :
        *((int *)((char *) deviceDist + ki * pitch) + nj);
    if (ni >= n || nj >= n) return;
    v = *((int *)((char *) deviceDist + ni * pitch) + nj);
    __syncthreads();
    #pragma unroll
    for (int k = 0; k < kmax; ++k) {
        if ((t = vertDist[i][k] + horzDist[k][j]) < v)
            v = t;
    }
    *((int *)((char *) deviceDist + ni * pitch) + nj) = v;
}

int main(int argc, char* argv[]) {
    int n, m, a, b, v, *Dist, *deviceDist;
    std::ifstream infile(argv[1]);
    infile >> n >> m;
    hipHostMalloc((void **) &Dist, n * n * sizeof(int), hipHostMallocDefault);
    for (size_t i = 0; i < n; ++i) {
        for (size_t j = 0; j < n; ++j) {
            Dist[i * n + j] = (i == j ? 0 : INF);
        }
    }
    while (m--) {
        infile >> a >> b >> v;
        Dist[a * n + b] = v;
    }
    infile.close();
    // blocked FW
    size_t pitch;
    hipMallocPitch(&deviceDist, &pitch, n * sizeof(int), n);
    hipMemcpy2D(deviceDist, pitch, Dist, n * sizeof(int), n * sizeof(int), n, hipMemcpyHostToDevice);
    const int B = atoi(argv[3]), round = ceil(n, B);
    dim3 thread(B, B);
    #pragma unroll
    for (int r = 0; r < round; ++r) {
        phase1<32><<<1, thread>>>(n, r, deviceDist, pitch);
        phase2<32><<<dim3(2, round), thread>>>(n, r, deviceDist, pitch);
        phase3<32><<<dim3(round, round), thread>>>(n, r, deviceDist, pitch);
    }
    hipMemcpy2D(Dist, n * sizeof(int), deviceDist, pitch, n * sizeof(int), n, hipMemcpyDeviceToHost);
    hipFree(deviceDist);
    // output
    std::ofstream outfile(argv[2], std::ofstream::binary);
    for (size_t i = 0; i < n; ++i) {
        for (size_t j = 0; j < n; ++j) {
            if (Dist[i * n + j] > INF)
                Dist[i * n + j] = INF;
        }
		outfile.write(reinterpret_cast<const char *>(Dist + i * n), sizeof(int) * n);
    }
    return 0;
}

