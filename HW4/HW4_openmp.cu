#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <omp.h>
#include <hip/hip_runtime.h>
#define INF 1000000000
#define V 20001
#define ceil(a, b) (1 + (((a) - 1) / (b)))
#define min(a, b) (((a) < (b)) ? (a) : (b))

// <<<1, dim3(B, B)>>>
template<int B>
__global__ void phase1(int n, int r, int *deviceDist) {
    const int kmin = r * B,
        kmax = min(B, n - kmin),
        i = threadIdx.y,
        j = threadIdx.x,
        ni = kmin + i,
        nj = kmin + j;
    int t, v;
    if (ni >= n || nj >= n) return;
    __shared__ int sharedDist[B][B];
    v = sharedDist[i][j] = *(deviceDist + ni * n + nj);
    #pragma unroll
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        if ((t = sharedDist[i][k] + sharedDist[k][j]) < v)
            v = sharedDist[i][j] = t;
    }
    *(deviceDist + ni * n + nj) = v;
}
// <<<dim3(2, round), dim3(B, B)>>>
template<int B>
__global__ void phase2(int n, int r, int *deviceDist) {
    if (blockIdx.y == r) return;
    const int kmin = r * B,
        kmax = min(B, n - kmin),
        i = threadIdx.y,
        j = threadIdx.x,
        ni = (blockIdx.x == 0 ? blockIdx.y : r) * B + i,
        nj = (blockIdx.x == 0 ? r : blockIdx.y) * B + j,
        ki = kmin + i,
        kj = kmin + j;
    __shared__ int sharedDist[B][B], pivotDist[B][B];
    int t, v,
        (*fromDist)[B] = blockIdx.x == 0 ? sharedDist : pivotDist,
        (*toDist)[B] = blockIdx.x == 0 ? pivotDist : sharedDist;
    v = sharedDist[i][j] = ni >= n || nj >= n ? INF : *(deviceDist + ni * n + nj);
    pivotDist[i][j] = ki >= n || kj >= n ? INF : *(deviceDist + ki * n + kj);
    if (ni >= n || nj >= n) return;
    #pragma unroll
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        if ((t = fromDist[i][k] + toDist[k][j]) < v)
            v = sharedDist[i][j] = t;
    }
    *(deviceDist + ni * n + nj) = v;
}
// <<<dim3(round, round), dim3(B, B)>>>
template<int B>
__global__ void phase3(int n, int r, int *deviceDist, int ompIdx) {
    if (blockIdx.x == r || blockIdx.y + ompIdx == r) return;
    const int kmin = r * B,
        kmax = min(B, n - kmin),
        i = threadIdx.y,
        j = threadIdx.x,
        ni = (blockIdx.y + ompIdx) * B + i,
        nj = blockIdx.x * B + j,
        ki = kmin + i,
        kj = kmin + j;
    __shared__ int vertDist[B][B], horzDist[B][B];
    int t, v;
    vertDist[i][j] = ni >= n || kj >= n ? INF : *(deviceDist + ni * n + kj);
    horzDist[i][j] = ki >= n || nj >= n ? INF : *(deviceDist + ki * n + nj);
    if (ni >= n || nj >= n) return;
    v = *(deviceDist + ni * n + nj);
    __syncthreads();
    #pragma unroll
    for (int k = 0; k < kmax; ++k) {
        if ((t = vertDist[i][k] + horzDist[k][j]) < v)
            v = t;
    }
    *(deviceDist + ni * n + nj) = v;
}

int main(int argc, char* argv[]) {
    int n, m, a, b, v, *Dist, *deviceDist[2];
    std::ifstream infile(argv[1]);
    infile >> n >> m;
    hipHostMalloc((void **) &Dist, n * n * sizeof(int), hipHostMallocDefault);
    for (size_t i = 0; i < n; ++i) {
        for (size_t j = 0; j < n; ++j) {
            Dist[i * n + j] = (i == j ? 0 : INF);
        }
    }
    while (m--) {
        infile >> a >> b >> v;
        Dist[a * n + b] = v;
    }
    infile.close();
    // blocked FW
    const int B = atoi(argv[3]), round = ceil(n, B),
            rhalf = round / 2;
    dim3 thread(B, B);
    #pragma unroll 2
    for (size_t i = 0 ; i < 2; ++i) {
        hipSetDevice(i);
        hipMalloc(&deviceDist[i], n * n * sizeof(int));
        hipMemcpy(deviceDist[i], Dist, n * n * sizeof(int), hipMemcpyHostToDevice);
    }
    #pragma omp parallel num_threads(2)
    {
        const int dev = omp_get_thread_num(),
            rblock = dev == 0 ? (round + 1) / 2 : rhalf;
        int kmin = 0, kmax;
        hipSetDevice(dev);
        phase1<32><<<1, thread>>>(n, 0, deviceDist[dev]);
        phase2<32><<<dim3(2, round), thread>>>(n, 0, deviceDist[dev]);
        phase3<32><<<dim3(round, rblock), thread>>>(n, 0, deviceDist[dev], dev == 0 ? rhalf : 0);
        for (int r = 1; r < round; ++r) {
            kmin += B, kmax = min(B, n - kmin);
            if (dev == 0) {
                hipMemcpyPeer(
                    deviceDist[1] + rhalf * B * n + kmin, 1,
                    deviceDist[0] + rhalf * B * n + kmin, 0,
                    kmax * (n - rhalf * B));
                if (r >= rhalf)
                    hipMemcpyPeer(
                        deviceDist[1] + kmin * n, 1,
                        deviceDist[0] + kmin * n, 0,
                        kmax * n * sizeof(int));
            } else if (rhalf) {
               hipMemcpyPeer(
                   deviceDist[0] + kmin, 0,
                   deviceDist[1] + kmin, 1,
                   kmax * rhalf * B);
               if (r < rhalf)
                  hipMemcpyPeer(
                      deviceDist[0] + kmin * n, 0,
                      deviceDist[1] + kmin * n, 1,
                      kmax * n * sizeof(int));
            }
            #pragma omp barrier
            phase1<32><<<1, thread>>>(n, r, deviceDist[dev]);
            phase2<32><<<dim3(2, round), thread>>>(n, r, deviceDist[dev]);
            phase3<32><<<dim3(round, rblock), thread>>>(n, r, deviceDist[dev], dev == 0 ? rhalf : 0);
        }
        hipDeviceSynchronize();
        if (dev == 0)
            hipMemcpy(
                Dist + rhalf * B * n, deviceDist[0] + rhalf * B * n,
                (n - rhalf * B) * n * sizeof(int), hipMemcpyDeviceToHost);
        else
            hipMemcpy(Dist, deviceDist[1], rhalf * B * n * sizeof(int), hipMemcpyDeviceToHost);
        hipFree(deviceDist[dev]);
        #pragma omp barrier
    }
    // output
    std::ofstream outfile(argv[2], std::ofstream::binary);
    outfile.write(reinterpret_cast<const char *>(Dist), n * n * sizeof(int));
    return 0;
}

